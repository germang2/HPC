
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// CUDA kernel
__global__ void multMatGPU(double *A, int rowsA, int colsA, double *B, int rowsB, int colsB, double *C){
   double sum = 0.0;
   int k;
   // Gets the id for the row and the col
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;
   
   // Multiplication, needs one cycle
   if(row < rowsA && col < colsB){
      sum = 0.0;
      for(k = 0; k < rowsB; k++){
         sum += A[(row * colsA) + k] * B[(k * colsB) + col];
       }
       C[row*colsB+col] = sum;
   }
}

__host__ void multMatHost(double *A, int rowsA, int colsA, double *B, int rowsB, int colsB, double *C){
   int k, i, j, index = 0;
   double sum = 0.0;
   for(i = 0; i < rowsA; i++){
      for(j = 0; j < colsB; j++){
         sum = 0.0;
         for(k = 0; k < rowsA; k++){
            sum += A[(i * colsA) + k] * B[(k * colsB) + j]; 
         }
         C[i*colsB+j] = sum;
         index += 1; 
      }
   }
}


int main(){
   double *h_A, *h_B, *h_C, *CPU, *d_A, *d_B, *d_C;
   int rowsA = 1000, colsA = 1000, rowsB = 1000, colsB = 1000;
   int i, j;   
   if (colsA != rowsA)
	return 1;

   h_A = (double*)malloc((rowsA*colsA)*sizeof(double));
   h_B = (double*)malloc((rowsB*colsB)*sizeof(double));
   CPU = (double*)malloc((rowsA*colsB)*sizeof(double));


   for(i = 0; i < rowsA; i++)
      for(j = 0; j < colsA; j++)
         h_A[i* colsA+j] = i+j;
   
   for(i = 0; i < rowsB; i++)
      for(j = 0; j < colsB; j++)
         h_B[i* colsB+j] = i+j;
   
   hipMalloc((void **)&d_A, rowsA*colsA);
   hipMalloc((void **)&d_B, rowsB*colsB);
   hipMalloc((void **)&d_C, rowsA*colsB); 
   
   hipMemcpy(d_A, h_A, rowsA*colsA, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, rowsA*colsB, hipMemcpyHostToDevice);

   int gridSize, blockSize;
   blockSize = 1024;
   gridSize = (int)ceil((float)colsB/blockSize);   

   multMatGPU<<<gridSize, blockSize>>>(d_A, rowsA, colsA, d_B, rowsB, colsB, d_C);
   hipMemcpy(h_C, d_C, rowsA*colsB, hipMemcpyDeviceToHost);
   multMatHost(h_A, rowsA, colsA, h_B, rowsB, colsB, CPU);

   int val = 1;
   for(i = 0; i < colsB; i++)
      if(CPU[i] != h_C[i])
         val = 0;

   if(val == 1)
      printf("Same results");
   else
      printf("Diferents results");

   free(h_A);
   free(h_B);
   free(h_C);
   free(CPU);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   return 0;
}


