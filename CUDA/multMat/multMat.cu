
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// CUDA kernel
__global__ void multMatGPU(double *A, int rowsA, int colsA, double *B, int rowsB, int colsB, double *C){
   double sum = 0.0;
   int k;
   // Gets the id for the row and the col
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;
   
   // Multiplication, needs one cycle
   if(row < rowsA && col < colsB){
      sum = 0.0;
      for(k = 0; k < rowsB; k++){
         sum += A[(row * colsA) + k] * B[(k * colsB) + col];
       }
       C[row*colsB+col] = sum;
   }
}

__host__ void multMatHost(double *A, int rowsA, int colsA, double *B, int rowsB, int colsB, double *C){
   int k, i, j, index = 0;
   double sum = 0.0;
   for(i = 0; i < rowsA; i++){
      for(j = 0; j < colsB; j++){
         sum = 0.0;
         for(k = 0; k < rowsA; k++){
            sum += A[(i * colsA) + k] * B[(k * colsB) + j]; 
         }
         C[i*colsB+j] = sum;
         index += 1; 
      }
   }
}


int main(){
   double *h_A, *h_B, *h_C, *CPU, *d_A, *d_B, *d_C;
   int rowsA = 1000, colsA = 1000, rowsB = 1000, colsB = 1000;
   int i, j;   
   if (colsA != rowsA)
	return 1;

   h_A = (double*)malloc((rowsA*colsA)*sizeof(double));
   h_B = (double*)malloc((rowsB*colsB)*sizeof(double));
   CPU = (double*)malloc((rowsA*colsB)*sizeof(double));
   h_C = (double*)malloc((rowsA*colsB)*sizeof(double));

   for(i = 0; i < rowsA; i++)
      for(j = 0; j < colsA; j++)
         h_A[i* colsA+j] = i+j;
   
   for(i = 0; i < rowsB; i++)
      for(j = 0; j < colsB; j++)
         h_B[i* colsB+j] = i+j;
   
   hipMalloc((void **)&d_A, rowsA*colsA*sizeof(double));
   hipMalloc((void **)&d_B, rowsB*colsB*sizeof(double));
   hipMalloc((void **)&d_C, rowsA*colsB*sizeof(double)); 
   
   hipMemcpy(d_A, h_A, rowsA*colsA*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, rowsA*colsB*sizeof(double), hipMemcpyHostToDevice);

   int threads = 32;
   dim3 gridSize(ceil(colsB/(double)threads), ceil(rowsA/(double)threads), 1);
   dim3 blockSize(threads, threads, 1);
  

   multMatGPU<<<gridSize, blockSize>>>(d_A, rowsA, colsA, d_B, rowsB, colsB, d_C);
   hipDeviceSynchronize();
   hipMemcpy(h_C, d_C, rowsA*colsB*sizeof(double), hipMemcpyDeviceToHost);
   multMatHost(h_A, rowsA, colsA, h_B, rowsB, colsB, CPU);

   int val = 1;
   for(i = 0; i < rowsA; i++){
      for(j = 0; j < colsB; j++){
         if(CPU[i*colsB+j] != h_C[i*colsB+j])
            val = 0;
      }
   }

   if(val == 1)
      printf("Same results");
   else
      printf("Diferents results");

   free(h_A);
   free(h_B);
   free(h_C);
   free(CPU);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   return 0;
}


